
#include <hip/hip_runtime.h>
#include <iostream>
#include <sys/time.h>

__global__ void saxpyDevice(int n, float a, float *x, float *y){
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    if (i < n) y[i] = a*x[i] + y[i];
}

void saxpy(int n, float a, float *x, float *y){
    float *d_x, *d_y;

    // allocate GPU memory, and upload data
    hipMalloc(&d_x, n*sizeof(float)); 
    hipMalloc(&d_y, n*sizeof(float));
    hipMemcpy(d_x, x, n*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, n*sizeof(float), hipMemcpyHostToDevice);

    // send instructions to GPU
    saxpyDevice<<<(n+255)/256, 256>>>(n, 2.0f, d_x, d_y);

    // download data, and free GPU memory
    hipMemcpy(y, d_y, n*sizeof(float), hipMemcpyDeviceToHost);
    hipFree(d_x);
    hipFree(d_y);
}

int main(void){
    int N = 1<<20;
    float *x, *y;
    x = (float*)malloc(N*sizeof(float));
    y = (float*)malloc(N*sizeof(float));
    struct timeval t0, t1;
    gettimeofday(&t0, NULL);
    for (int i=0; i<100; i++)
        saxpy(N, 2.0f, x, y);
    gettimeofday(&t1, NULL);
    std::cout<<"CUDA = "<<(t1.tv_sec - t0.tv_sec)*1000 + (t1.tv_usec-t0.tv_usec)/1000<<"ms"<<std::endl;
	return 0;
}
